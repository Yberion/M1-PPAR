#include "hip/hip_runtime.h"

// GPU kernel
// data_size = data_size_per_thread
__global__ void summation_kernel(int data_size, float* data_out)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	float res = 0.0F;
	int op = -1;
	for(int j = ind*data_size; j < (ind+1)*data_size; j++){
		res += j == 0 ? 0 : (float) 1/j * op;
		op *= -1;
	}
	data_out[ind] = res;
}

// GPU kernel
__global__ void summation_kernel_2(int data_size, float* data_out)
{
	int threadNumber = blockIdx.x * blockDim.x + threadIdx.x;

	int op;
	float res = 0.0F;

	for (int i = 0; i < data_size: ++i)
	{
		op = (i % 2 == 0) ? 1 : -1;

		res = (i == 0) ? 0 : (float) 1 / (i * threadNumber) * op;
	}

	data_out[threadNumber] = res;
}