#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdlib.h>

#include "summation_kernel.cu"

/*
// CPU implementation
float log2_series_brandon(int n)
{
	float result = 0.0F;

	for (int i = 0; i < n; ++i)
	{
		result += (powf(-1, i)) / (i + 1);
	}
	
	return result;
}
*/

/*
// CPU implementation
float log2_series_thomas(int n)
{
	float res = 0.0F;
	
    int op = 1;
	
    for(int i=1; i<=n; i++)
	{
        res += (float) 1/i * op;
        op *= -1;
    }
	
	return res;
}
*/

// CPU implementation
float log2_series(int n)
{
	float res = 0.0F;
	
    int op = 1;
	
    for(int i=1; i<=n; i++)
	{
        res += (float) 1/i * op;
        op *= -1;
    }
	
	return res;
}

int main(int argc, char ** argv)
{
    int data_size = 1024 * 1024 * 128;

    // Run CPU version
    double start_time = getclock();
    float log2 = log2_series(data_size);
    double end_time = getclock();
    
    printf("CPU result: %f\n", log2);
    printf(" log(2)=%f\n", log(2.0));
    printf(" time=%fs\n", end_time - start_time);
    
    // Parameter definition
    int threads_per_block = 4 * 32;
    int blocks_in_grid = 8;
    
    int num_threads = threads_per_block * blocks_in_grid;

    // Timer initialization
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

    int results_size = num_threads;
    int alloc_size = results_size * sizeof(float);

    // Allocating output data on CPU
	// TODO
    float* data_out_cpu = (float *)data_out_cpu = calloc(results_size, sizeof(float));

	// Allocating output data on GPU
    // TODO
    float* data_out_gpu;
    int* data_size_gpu;
    int data_size_per_thread = data_size / num_threads;

    hipMalloc((void **)&data_out_gpu, alloc_size);
    hipMalloc((void **)&data_size_gpu, sizeof(int));

    hipMemset((void *)data_out_gpu, 0, alloc_size);
    hipMemset((void *)data_size_gpu, 0, sizeof(int));

    hipMemcpy(data_size_gpu, &data_size_per_thread, sizeof(int), hipMemcpyHostToDevice);

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    // Execute kernel
    // TODO

    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    // Get results back
    // TODO
    hipMemcpy(data_out_cpu, data_out_gpu, alloc_size, hipMemcpyDeviceToHost);
    
    // Finish reduction
    // TODO
	float sum = 0.0F;

    for (int i = 0; i < results_size; ++i)
    {
        sum += data_out_cpu[i];
    }
    
    // Cleanup
    // TODO
    hipFree(data_size_gpu);
    hipFree(data_out_gpu);
    free(data_out_cpu);
    
    printf("GPU results:\n");
    printf(" Sum: %f\n", sum);
    
    float elapsedTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms

    double total_time = elapsedTime / 1000.;	// s
    double time_per_iter = total_time / (double)data_size;
    double bandwidth = sizeof(float) / time_per_iter; // B/s
    
    printf(" Total time: %g s,\n Per iteration: %g ns\n Throughput: %g GB/s\n",
    	total_time,
    	time_per_iter * 1.e9,
    	bandwidth / 1.e9);
  
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
    return 0;
}

